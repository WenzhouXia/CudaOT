#include"image_ot_solver.h"
#include"Common/handler_cuda_error.h"
#include<hip/hip_runtime_api.h>
#include <stdio.h>
#include<hipblas.h>
#include<Common.h>
#include"family.h"
#include <iostream>
#include<SparseSinkhorn/solver.cu>
#include"layer_manage.h"
#include"memory_manage.h"
#include"data_struct.h"
#include"kernel_manage.h"
#include"eps_handler.h"


double image_ot_solver(int image_size,
                     std::vector<double> &muXdat,
                     std::vector<double> &muYdat, int dev, double maxError_input, double truncation_threshold){

    double res = 0.0;
    switch (image_size) {
    case 32:
        res = cuda_image_ot_solver<32>(muXdat,muYdat, dev, maxError_input, truncation_threshold);
        break;
    case 64:
        res = cuda_image_ot_solver<64>(muXdat,muYdat, dev, maxError_input, truncation_threshold);
        break;
    case 128:
        res = cuda_image_ot_solver<128>(muXdat,muYdat, dev, maxError_input, truncation_threshold);
        break;
    case 256:
        res = cuda_image_ot_solver<256>(muXdat,muYdat, dev, maxError_input, truncation_threshold);
        break;
    case 512:
        res = cuda_image_ot_solver<512>(muXdat,muYdat, dev, maxError_input, truncation_threshold);
        break;
    default:
        break;
    }
    return res;

}

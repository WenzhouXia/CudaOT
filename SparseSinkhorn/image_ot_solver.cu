#include"image_ot_solver.h"
#include"Common/handler_cuda_error.h"
#include<hip/hip_runtime_api.h>
#include <stdio.h>
#include<hipblas.h>
#include<Common.h>
#include"family.h"
#include <iostream>
#include<SparseSinkhorn/solver.cu>
#include"layer_manage.h"
#include"memory_manage.h"
#include"data_struct.h"
#include"kernel_manage.h"
#include"eps_handler.h"


double image_ot_solver(int image_size,
                     std::vector<double> &muXdat,
                     std::vector<double> &muYdat){

    double res = 0.0;
    switch (image_size) {
    case 32:
        res = cuda_image_ot_solver<32>(muXdat,muYdat);
        break;
    case 64:
        res = cuda_image_ot_solver<64>(muXdat,muYdat);
        break;
    case 128:
        res = cuda_image_ot_solver<128>(muXdat,muYdat);
        break;
    case 256:
        res = cuda_image_ot_solver<256>(muXdat,muYdat);
        break;
    case 512:
        res = cuda_image_ot_solver<512>(muXdat,muYdat);
        break;
    default:
        break;
    }
    return res;

}
